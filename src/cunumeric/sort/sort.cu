#include "hip/hip_runtime.h"
/* Copyright 2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/sort/sort.h"
#include "cunumeric/sort/sort_template.inl"
#include "cunumeric/utilities/thrust_allocator.h"

#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/binary_search.h>
#include <thrust/tuple.h>
#include <thrust/execution_policy.h>
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_segmented_radix_sort.cuh>
#include <cub/thread/thread_search.cuh>

#include "cunumeric/cuda_help.h"

// above this threshold segment sort will be performed
// by hipcub::DeviceSegmentedRadixSort instead of thrust::(stable_)sort
// with tuple keys (not available for complex)
#define SEGMENT_THRESHOLD_RADIX_SORT 400

namespace cunumeric {

using namespace Legion;

struct multiply : public thrust::unary_function<size_t, size_t> {
  const size_t constant;

  multiply(size_t _constant) : constant(_constant) {}

  __host__ __device__ size_t operator()(const size_t& input) const { return input * constant; }
};

struct modulusWithOffset {
  const size_t constant;

  modulusWithOffset(size_t _constant) : constant(_constant) {}

  __host__ __device__ int64_t operator()(const int64_t& lhs, const int64_t& rhs) const
  {
    return lhs % rhs + constant;
  }
};

template <class VAL>
void cub_local_sort(const VAL* values_in,
                    VAL* values_out,
                    const int64_t* indices_in,
                    int64_t* indices_out,
                    const size_t volume,
                    const size_t sort_dim_size,
                    hipStream_t stream)
{
  Buffer<VAL> keys_in;
  const VAL* values_in_cub = values_in;
  if (values_in == values_out) {
    keys_in       = create_buffer<VAL>(volume, Legion::Memory::Kind::GPU_FB_MEM);
    values_in_cub = keys_in.ptr(0);
    CHECK_CUDA(hipMemcpyAsync(
      keys_in.ptr(0), values_out, sizeof(VAL) * volume, hipMemcpyDeviceToDevice, stream));
  }

  size_t temp_storage_bytes = 0;
  if (indices_out == nullptr) {
    if (volume == sort_dim_size) {
      // sort (initial call to compute buffer size)
      hipcub::DeviceRadixSort::SortKeys(
        nullptr, temp_storage_bytes, values_in_cub, values_out, volume, 0, sizeof(VAL) * 8, stream);
      auto temp_storage =
        create_buffer<unsigned char>(temp_storage_bytes, Legion::Memory::Kind::GPU_FB_MEM);
      hipcub::DeviceRadixSort::SortKeys(temp_storage.ptr(0),
                                     temp_storage_bytes,
                                     values_in_cub,
                                     values_out,
                                     volume,
                                     0,
                                     sizeof(VAL) * 8,
                                     stream);
      temp_storage.destroy();
    } else {
      // segmented sort (initial call to compute buffer size)
      // generate start/end positions for all segments via iterators to avoid allocating buffers
      auto off_start_pos_it = thrust::make_transform_iterator(
        thrust::make_counting_iterator<size_t>(0), multiply(sort_dim_size));
      auto off_end_pos_it = thrust::make_transform_iterator(
        thrust::make_counting_iterator<size_t>(1), multiply(sort_dim_size));

      hipcub::DeviceSegmentedRadixSort::SortKeys(nullptr,
                                              temp_storage_bytes,
                                              values_in_cub,
                                              values_out,
                                              volume,
                                              volume / sort_dim_size,
                                              off_start_pos_it,
                                              off_end_pos_it,
                                              0,
                                              sizeof(VAL) * 8,
                                              stream);
      auto temp_storage =
        create_buffer<unsigned char>(temp_storage_bytes, Legion::Memory::Kind::GPU_FB_MEM);

      hipcub::DeviceSegmentedRadixSort::SortKeys(temp_storage.ptr(0),
                                              temp_storage_bytes,
                                              values_in_cub,
                                              values_out,
                                              volume,
                                              volume / sort_dim_size,
                                              off_start_pos_it,
                                              off_end_pos_it,
                                              0,
                                              sizeof(VAL) * 8,
                                              stream);
      temp_storage.destroy();
    }
  } else {
    Buffer<int64_t> idx_in;
    const int64_t* indices_in_cub = indices_in;
    if (indices_in == indices_out) {
      idx_in         = create_buffer<int64_t>(volume, Legion::Memory::Kind::GPU_FB_MEM);
      indices_in_cub = idx_in.ptr(0);
      CHECK_CUDA(hipMemcpyAsync(
        idx_in.ptr(0), indices_out, sizeof(int64_t) * volume, hipMemcpyDeviceToDevice, stream));
    }

    if (volume == sort_dim_size) {
      // argsort (initial call to compute buffer size)
      hipcub::DeviceRadixSort::SortPairs(nullptr,
                                      temp_storage_bytes,
                                      values_in_cub,
                                      values_out,
                                      indices_in_cub,
                                      indices_out,
                                      volume,
                                      0,
                                      sizeof(VAL) * 8,
                                      stream);

      auto temp_storage =
        create_buffer<unsigned char>(temp_storage_bytes, Legion::Memory::Kind::GPU_FB_MEM);

      hipcub::DeviceRadixSort::SortPairs(temp_storage.ptr(0),
                                      temp_storage_bytes,
                                      values_in_cub,
                                      values_out,
                                      indices_in_cub,
                                      indices_out,
                                      volume,
                                      0,
                                      sizeof(VAL) * 8,
                                      stream);
      temp_storage.destroy();
    } else {
      // segmented argsort (initial call to compute buffer size)
      // generate start/end positions for all segments via iterators to avoid allocating buffers
      auto off_start_pos_it = thrust::make_transform_iterator(
        thrust::make_counting_iterator<size_t>(0), multiply(sort_dim_size));
      auto off_end_pos_it = thrust::make_transform_iterator(
        thrust::make_counting_iterator<size_t>(1), multiply(sort_dim_size));

      hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr,
                                               temp_storage_bytes,
                                               values_in_cub,
                                               values_out,
                                               indices_in_cub,
                                               indices_out,
                                               volume,
                                               volume / sort_dim_size,
                                               off_start_pos_it,
                                               off_end_pos_it,
                                               0,
                                               sizeof(VAL) * 8,
                                               stream);

      auto temp_storage =
        create_buffer<unsigned char>(temp_storage_bytes, Legion::Memory::Kind::GPU_FB_MEM);

      hipcub::DeviceSegmentedRadixSort::SortPairs(temp_storage.ptr(0),
                                               temp_storage_bytes,
                                               values_in_cub,
                                               values_out,
                                               indices_in_cub,
                                               indices_out,
                                               volume,
                                               volume / sort_dim_size,
                                               off_start_pos_it,
                                               off_end_pos_it,
                                               0,
                                               sizeof(VAL) * 8,
                                               stream);
      temp_storage.destroy();
    }
    if (indices_in == indices_out) idx_in.destroy();
  }

  if (values_in == values_out) keys_in.destroy();
}

template <class VAL>
void thrust_local_sort(const VAL* values_in,
                       VAL* values_out,
                       const int64_t* indices_in,
                       int64_t* indices_out,
                       const size_t volume,
                       const size_t sort_dim_size,
                       const bool stable,
                       hipStream_t stream)
{
  auto alloc       = ThrustAllocator(Memory::GPU_FB_MEM);
  auto exec_policy = thrust::cuda::par(alloc).on(stream);

  if (values_in != values_out) {
    // not in-place --> need a copy
    CHECK_CUDA(hipMemcpyAsync(
      values_out, values_in, sizeof(VAL) * volume, hipMemcpyDeviceToDevice, stream));
  }
  if (indices_in != indices_out) {
    // not in-place --> need a copy
    CHECK_CUDA(hipMemcpyAsync(
      indices_out, values_in, sizeof(int64_t) * volume, hipMemcpyDeviceToDevice, stream));
  }

  if (indices_out == nullptr) {
    if (volume == sort_dim_size) {
      if (stable) {
        thrust::stable_sort(exec_policy, values_out, values_out + volume);
      } else {
        thrust::sort(exec_policy, values_out, values_out + volume);
      }
    } else {
      auto sort_id = create_buffer<uint64_t>(volume, Legion::Memory::Kind::GPU_FB_MEM);
      // init combined keys
      thrust::transform(exec_policy,
                        thrust::make_counting_iterator<uint64_t>(0),
                        thrust::make_counting_iterator<uint64_t>(volume),
                        thrust::make_constant_iterator<uint64_t>(sort_dim_size),
                        sort_id.ptr(0),
                        thrust::divides<uint64_t>());
      auto combined = thrust::make_zip_iterator(thrust::make_tuple(sort_id.ptr(0), values_out));

      if (stable) {
        thrust::stable_sort(
          exec_policy, combined, combined + volume, thrust::less<thrust::tuple<size_t, VAL>>());
      } else {
        thrust::sort(
          exec_policy, combined, combined + volume, thrust::less<thrust::tuple<size_t, VAL>>());
      }

      sort_id.destroy();
    }
  } else {
    if (volume == sort_dim_size) {
      if (stable) {
        thrust::stable_sort_by_key(exec_policy, values_out, values_out + volume, indices_out);
      } else {
        thrust::sort_by_key(exec_policy, values_out, values_out + volume, indices_out);
      }
    } else {
      auto sort_id = create_buffer<uint64_t>(volume, Legion::Memory::Kind::GPU_FB_MEM);
      // init combined keys
      thrust::transform(exec_policy,
                        thrust::make_counting_iterator<uint64_t>(0),
                        thrust::make_counting_iterator<uint64_t>(volume),
                        thrust::make_constant_iterator<uint64_t>(sort_dim_size),
                        sort_id.ptr(0),
                        thrust::divides<uint64_t>());
      auto combined = thrust::make_zip_iterator(thrust::make_tuple(sort_id.ptr(0), values_out));

      if (stable) {
        thrust::stable_sort_by_key(exec_policy,
                                   combined,
                                   combined + volume,
                                   indices_out,
                                   thrust::less<thrust::tuple<size_t, VAL>>());
      } else {
        thrust::sort_by_key(exec_policy,
                            combined,
                            combined + volume,
                            indices_out,
                            thrust::less<thrust::tuple<size_t, VAL>>());
      }

      sort_id.destroy();
    }
  }
}

template <LegateTypeCode CODE>
struct support_cub : std::true_type {
};
template <>
struct support_cub<LegateTypeCode::COMPLEX64_LT> : std::false_type {
};
template <>
struct support_cub<LegateTypeCode::COMPLEX128_LT> : std::false_type {
};

template <LegateTypeCode CODE, std::enable_if_t<support_cub<CODE>::value>* = nullptr>
void local_sort(const legate_type_of<CODE>* values_in,
                legate_type_of<CODE>* values_out,
                const int64_t* indices_in,
                int64_t* indices_out,
                const size_t volume,
                const size_t sort_dim_size,
                const bool stable,  // cub sort is always stable
                hipStream_t stream)
{
  using VAL = legate_type_of<CODE>;
  // fallback to thrust approach as segmented radix sort is not suited for small segments
  if (volume == sort_dim_size || sort_dim_size > SEGMENT_THRESHOLD_RADIX_SORT) {
    cub_local_sort<VAL>(
      values_in, values_out, indices_in, indices_out, volume, sort_dim_size, stream);
  } else {
    thrust_local_sort<VAL>(
      values_in, values_out, indices_in, indices_out, volume, sort_dim_size, stable, stream);
  }
}

template <LegateTypeCode CODE, std::enable_if_t<!support_cub<CODE>::value>* = nullptr>
void local_sort(const legate_type_of<CODE>* values_in,
                legate_type_of<CODE>* values_out,
                const int64_t* indices_in,
                int64_t* indices_out,
                const size_t volume,
                const size_t sort_dim_size,
                const bool stable,
                hipStream_t stream)
{
  using VAL = legate_type_of<CODE>;
  thrust_local_sort<VAL>(
    values_in, values_out, indices_in, indices_out, volume, sort_dim_size, stable, stream);
}

// auto align to multiples of 16 bytes
auto get_16b_aligned = [](auto bytes) { return std::max<size_t>(16, (bytes + 15) / 16 * 16); };
auto get_16b_aligned_count = [](auto count, auto element_bytes) {
  return (get_16b_aligned(count * element_bytes) + element_bytes - 1) / element_bytes;
};

template <typename VAL>
struct SortPiece {
  Buffer<VAL> values;
  Buffer<int64_t> indices;
  size_t size;
};

template <typename VAL>
struct Sample {
  VAL value;
  int32_t rank;
  size_t position;
};

template <typename VAL>
struct SampleComparator : public thrust::binary_function<Sample<VAL>, Sample<VAL>, bool> {
  __host__ __device__ bool operator()(const Sample<VAL>& lhs, const Sample<VAL>& rhs) const
  {
    // special case for unused samples
    if (lhs.rank < 0 || rhs.rank < 0) { return rhs.rank < 0 && lhs.rank >= 0; }

    if (lhs.value != rhs.value) {
      return lhs.value < rhs.value;
    } else if (lhs.rank != rhs.rank) {
      return lhs.rank < rhs.rank;
    } else {
      return lhs.position < rhs.position;
    }
  }
};

template <typename VAL>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  extract_samples(const VAL* data,
                  const size_t volume,
                  Sample<VAL>* samples,
                  const size_t num_local_samples,
                  const Sample<VAL> init_sample,
                  const size_t offset,
                  const size_t rank)
{
  const size_t sample_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (sample_idx >= num_local_samples) return;

  if (num_local_samples < volume) {
    const size_t index                    = (sample_idx + 1) * volume / num_local_samples - 1;
    samples[offset + sample_idx].value    = data[index];
    samples[offset + sample_idx].rank     = rank;
    samples[offset + sample_idx].position = index;
  } else {
    // edge case where num_local_samples > volume
    if (sample_idx < volume) {
      samples[offset + sample_idx].value    = data[sample_idx];
      samples[offset + sample_idx].rank     = rank;
      samples[offset + sample_idx].position = sample_idx;
    } else {
      samples[offset + sample_idx] = init_sample;
    }
  }
}

template <typename VAL>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  extract_split_positions(const VAL* data,
                          const size_t volume,
                          const Sample<VAL>* samples,
                          const size_t num_samples,
                          size_t* split_positions,
                          const size_t num_splitters,
                          const size_t rank)
{
  const size_t splitter_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (splitter_idx >= num_splitters) return;

  const size_t index         = (splitter_idx + 1) * num_samples / (num_splitters + 1) - 1;
  const Sample<VAL> splitter = samples[index];

  // now perform search on data to receive position *after* last element to be
  // part of the package for rank splitter_idx
  if (rank > splitter.rank) {
    // position of the last position with smaller value than splitter.value + 1
    split_positions[splitter_idx] = hipcub::LowerBound(data, volume, splitter.value);
  } else if (rank < splitter.rank) {
    // position of the first position with value larger than splitter.value
    split_positions[splitter_idx] = hipcub::UpperBound(data, volume, splitter.value);
  } else {
    split_positions[splitter_idx] = splitter.position + 1;
  }
}

template <typename VAL>
static SortPiece<VAL> sample_sort_nccl(SortPiece<VAL> local_sorted,
                                       size_t my_rank,
                                       size_t num_ranks,
                                       bool argsort,
                                       hipStream_t stream,
                                       ncclComm_t* comm)
{
  size_t volume = local_sorted.size;

  // collect local samples - for now we take num_ranks samples for every node
  // worst case this leads to 2*N/ranks elements on a single node
  size_t num_local_samples = num_ranks;

  size_t num_global_samples = num_local_samples * num_ranks;
  auto samples              = create_buffer<Sample<VAL>>(num_global_samples, Memory::GPU_FB_MEM);

  Sample<VAL> init_sample;
  {
    const size_t num_blocks = (num_local_samples + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    init_sample.rank        = -1;  // init samples that are not populated
    size_t offset           = num_local_samples * my_rank;
    extract_samples<<<num_blocks, THREADS_PER_BLOCK, 0, stream>>>(local_sorted.values.ptr(0),
                                                                  volume,
                                                                  samples.ptr(0),
                                                                  num_local_samples,
                                                                  init_sample,
                                                                  offset,
                                                                  my_rank);
  }

  // AllGather: check alignment? as we want to receive data in-place we take exact size for now
  CHECK_NCCL(ncclAllGather(samples.ptr(my_rank * num_ranks),
                           samples.ptr(0),
                           num_ranks * sizeof(Sample<VAL>),
                           ncclInt8,
                           *comm,
                           stream));

  // sort samples on device
  auto alloc       = ThrustAllocator(Memory::GPU_FB_MEM);
  auto exec_policy = thrust::cuda::par(alloc).on(stream);
  thrust::stable_sort(
    exec_policy, samples.ptr(0), samples.ptr(0) + num_global_samples, SampleComparator<VAL>());

  auto lower_bound          = thrust::lower_bound(exec_policy,
                                         samples.ptr(0),
                                         samples.ptr(0) + num_global_samples,
                                         init_sample,
                                         SampleComparator<VAL>());
  size_t num_usable_samples = lower_bound - samples.ptr(0);

  // select splitters / positions based on samples (on device)
  const size_t num_splitters = num_ranks - 1;
  auto split_positions       = create_buffer<size_t>(num_splitters, Memory::Z_COPY_MEM);
  {
    const size_t num_blocks = (num_splitters + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    extract_split_positions<<<num_blocks, THREADS_PER_BLOCK, 0, stream>>>(
      local_sorted.values.ptr(0),
      volume,
      samples.ptr(0),
      num_usable_samples,
      split_positions.ptr(0),
      num_splitters,
      my_rank);
  }

  // need to sync as we share values in between host/device
  CHECK_CUDA(hipStreamSynchronize(stream));

  // collect sizes2send, send to rank i: local_sort_data from positions  split_positions[i-1],
  // split_positions[i] - 1
  auto size_send = create_buffer<size_t>(num_ranks, Memory::Z_COPY_MEM);
  {
    size_t last_position = 0;
    for (size_t rank = 0; rank < num_ranks - 1; ++rank) {
      size_t cur_position = split_positions[rank];
      size_send[rank]     = cur_position - last_position;
      last_position       = cur_position;
    }
    size_send[num_ranks - 1] = volume - last_position;
  }

  // cleanup intermediate data structures
  samples.destroy();
  split_positions.destroy();

  // all2all exchange send/receive sizes
  auto size_recv = create_buffer<size_t>(num_ranks, Memory::Z_COPY_MEM);
  CHECK_NCCL(ncclGroupStart());
  for (size_t r = 0; r < num_ranks; r++) {
    CHECK_NCCL(ncclSend(size_send.ptr(r), 1, ncclUint64, r, *comm, stream));
    CHECK_NCCL(ncclRecv(size_recv.ptr(r), 1, ncclUint64, r, *comm, stream));
  }
  CHECK_NCCL(ncclGroupEnd());

  // need to sync as we share values in between host/device
  CHECK_CUDA(hipStreamSynchronize(stream));

  // handle alignment
  std::vector<size_t> aligned_pos_vals_send(num_ranks);
  std::vector<size_t> aligned_pos_idcs_send(num_ranks);
  size_t buf_size_send_vals_total = 0;
  size_t buf_size_send_idcs_total = 0;
  for (size_t i = 0; i < num_ranks; ++i) {
    // align buffer to allow data transfer of 16byte blocks
    aligned_pos_vals_send[i] = buf_size_send_vals_total;
    buf_size_send_vals_total += get_16b_aligned_count(size_send[i], sizeof(VAL));
    if (argsort) {
      aligned_pos_idcs_send[i] = buf_size_send_idcs_total;
      buf_size_send_idcs_total += get_16b_aligned_count(size_send[i], sizeof(int64_t));
    }
  }

  // copy values into aligned send buffer
  auto val_send_buf = local_sorted.values;
  if (buf_size_send_vals_total > volume) {
    val_send_buf = create_buffer<VAL>(buf_size_send_vals_total, Memory::GPU_FB_MEM);
    size_t pos   = 0;
    for (size_t r = 0; r < num_ranks; ++r) {
      CHECK_CUDA(hipMemcpyAsync(val_send_buf.ptr(aligned_pos_vals_send[r]),
                                 local_sorted.values.ptr(pos),
                                 sizeof(VAL) * size_send[r],
                                 hipMemcpyDeviceToDevice,
                                 stream));
      pos += size_send[r];
    }
    local_sorted.values.destroy();
  }

  // copy indices into aligned send buffer
  auto idc_send_buf = local_sorted.indices;
  if (argsort && buf_size_send_idcs_total > volume) {
    idc_send_buf = create_buffer<int64_t>(buf_size_send_idcs_total, Memory::GPU_FB_MEM);
    size_t pos   = 0;
    for (size_t r = 0; r < num_ranks; ++r) {
      CHECK_CUDA(hipMemcpyAsync(idc_send_buf.ptr(aligned_pos_idcs_send[r]),
                                 local_sorted.indices.ptr(pos),
                                 sizeof(int64_t) * size_send[r],
                                 hipMemcpyDeviceToDevice,
                                 stream));
      pos += size_send[r];
    }
    local_sorted.indices.destroy();
  }

  // allocate target buffers
  std::vector<SortPiece<VAL>> merge_buffers(num_ranks);
  for (size_t i = 0; i < num_ranks; ++i) {
    auto buf_size_vals_recv = get_16b_aligned_count(size_recv[i], sizeof(VAL));
    merge_buffers[i].values = create_buffer<VAL>(buf_size_vals_recv, Memory::GPU_FB_MEM);
    merge_buffers[i].size   = size_recv[i];
    if (argsort) {
      auto buf_size_idcs_recv  = get_16b_aligned_count(size_recv[i], sizeof(int64_t));
      merge_buffers[i].indices = create_buffer<int64_t>(buf_size_idcs_recv, Memory::GPU_FB_MEM);
    } else {
      merge_buffers[i].indices = create_buffer<int64_t>(0, Memory::GPU_FB_MEM);
    }
  }
  CHECK_NCCL(ncclGroupStart());
  for (size_t r = 0; r < num_ranks; r++) {
    CHECK_NCCL(ncclSend(val_send_buf.ptr(aligned_pos_vals_send[r]),
                        get_16b_aligned(size_send[r] * sizeof(VAL)),
                        ncclInt8,
                        r,
                        *comm,
                        stream));
    CHECK_NCCL(ncclRecv(merge_buffers[r].values.ptr(0),
                        get_16b_aligned(size_recv[r] * sizeof(VAL)),
                        ncclInt8,
                        r,
                        *comm,
                        stream));
  }
  CHECK_NCCL(ncclGroupEnd());

  if (argsort) {
    CHECK_NCCL(ncclGroupStart());
    for (size_t r = 0; r < num_ranks; r++) {
      CHECK_NCCL(ncclSend(idc_send_buf.ptr(aligned_pos_idcs_send[r]),
                          get_16b_aligned_count(size_send[r], sizeof(int64_t)),
                          ncclInt64,
                          r,
                          *comm,
                          stream));
      CHECK_NCCL(ncclRecv(merge_buffers[r].indices.ptr(0),
                          get_16b_aligned_count(size_recv[r], sizeof(int64_t)),
                          ncclInt64,
                          r,
                          *comm,
                          stream));
    }
    CHECK_NCCL(ncclGroupEnd());
  }

  // cleanup remaining buffers
  size_send.destroy();
  size_recv.destroy();
  val_send_buf.destroy();
  idc_send_buf.destroy();

  // now merge sort all into the result buffer
  // maybe k-way merge is more efficient here...
  for (size_t stride = 1; stride < num_ranks; stride *= 2) {
    for (size_t pos = 0; pos + stride < num_ranks; pos += 2 * stride) {
      SortPiece<VAL> source1 = merge_buffers[pos];
      SortPiece<VAL> source2 = merge_buffers[pos + stride];
      auto merged_size       = source1.size + source2.size;
      auto merged_values     = create_buffer<VAL>(merged_size);
      auto merged_indices    = source1.indices;  // will be overriden for argsort
      auto p_merged_values   = merged_values.ptr(0);
      auto p_values1         = source1.values.ptr(0);
      auto p_values2         = source2.values.ptr(0);
      if (argsort) {
        merged_indices = create_buffer<int64_t>(merged_size);
        // merge with key/value
        auto p_indices1       = source1.indices.ptr(0);
        auto p_indices2       = source2.indices.ptr(0);
        auto p_merged_indices = merged_indices.ptr(0);
        thrust::merge_by_key(exec_policy,
                             p_values1,
                             p_values1 + source1.size,
                             p_values2,
                             p_values2 + source2.size,
                             p_indices1,
                             p_indices2,
                             p_merged_values,
                             p_merged_indices);
        source1.indices.destroy();
      } else {
        thrust::merge(exec_policy,
                      p_values1,
                      p_values1 + source1.size,
                      p_values2,
                      p_values2 + source2.size,
                      p_merged_values);
      }

      source1.values.destroy();
      source2.values.destroy();
      source2.indices.destroy();

      merge_buffers[pos].values  = merged_values;
      merge_buffers[pos].indices = merged_indices;
      merge_buffers[pos].size    = merged_size;
    }
  }
  return merge_buffers[0];
}

/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename VAL>
struct SegmentSample {
  VAL value;
  size_t segment;
  int32_t rank;
  size_t position;
};

template <typename VAL>
struct SegmentMergePiece {
  Buffer<size_t> segments;
  Buffer<VAL> values;
  Buffer<int64_t> indices;
  size_t size;
};

template <typename VAL>
struct SegmentSampleComparator
  : public thrust::binary_function<SegmentSample<VAL>, SegmentSample<VAL>, bool> {
  __host__ __device__ bool operator()(const SegmentSample<VAL>& lhs,
                                      const SegmentSample<VAL>& rhs) const
  {
    if (lhs.segment != rhs.segment) {
      return lhs.segment < rhs.segment;
    } else {
      // special case for unused samples
      if (lhs.rank < 0 || rhs.rank < 0) { return rhs.rank < 0 && lhs.rank >= 0; }

      if (lhs.value != rhs.value) {
        return lhs.value < rhs.value;
      } else if (lhs.rank != rhs.rank) {
        return lhs.rank < rhs.rank;
      } else {
        return lhs.position < rhs.position;
      }
    }
  }
};

template <typename VAL>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  extract_split_positions_segments(const VAL* data,
                                   const size_t segment_size_l,
                                   const SegmentSample<VAL>* samples,
                                   const size_t num_segments_l,
                                   const size_t num_samples_per_segment,
                                   const size_t num_usable_samples_per_segment,
                                   size_t* split_positions,
                                   const size_t num_splitters,
                                   const size_t my_sort_rank)
{
  const size_t splitter_idx_g = blockIdx.x * blockDim.x + threadIdx.x;
  if (splitter_idx_g >= num_splitters) return;

  const size_t num_splitters_per_segment = num_splitters / num_segments_l;
  const size_t splitter_pos              = splitter_idx_g % num_splitters_per_segment;
  const size_t splitter_segment          = splitter_idx_g / num_splitters_per_segment;

  const size_t index =
    (splitter_pos + 1) * num_usable_samples_per_segment / (num_splitters_per_segment + 1) - 1;
  const SegmentSample<VAL> splitter = samples[splitter_segment * num_samples_per_segment + index];

  // now perform search on data to receive position *after* last element to be
  // part of the package for my_sort_rank splitter_idx_g
  const size_t offset = splitter_segment * segment_size_l;
  if (my_sort_rank > splitter.rank) {
    // position of the last position with smaller value than splitter.value + 1
    split_positions[splitter_idx_g] =
      hipcub::LowerBound(data + offset, segment_size_l, splitter.value) + offset;
  } else if (my_sort_rank < splitter.rank) {
    // position of the first position with value larger than splitter.value
    split_positions[splitter_idx_g] =
      hipcub::UpperBound(data + offset, segment_size_l, splitter.value) + offset;
  } else {
    split_positions[splitter_idx_g] = splitter.position + 1;
  }
}

__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  initialize_segment_start_positions(const size_t* start_positions,
                                     const size_t num_segments_l,
                                     size_t* segment_ids,
                                     const size_t num_segment_ids)
{
  const size_t segment_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (segment_idx >= num_segments_l) return;

  unsigned long long int* ptr = (unsigned long long int*)segment_ids;

  const size_t position = start_positions[segment_idx];
  if (position < num_segment_ids) atomicAdd(&(ptr[position]), (unsigned long long int)1l);
}

__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  extract_segment_sizes(const size_t* segments,
                        const size_t size,
                        int64_t* segments_diff,
                        const size_t num_segments_l,
                        const size_t segments_size_l)
{
  const size_t segment_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (segment_idx >= num_segments_l) return;

  const size_t position = hipcub::LowerBound(segments, size, segment_idx);
  const size_t next_position =
    hipcub::LowerBound(segments + position, size - position, segment_idx + 1) + position;

  const size_t segment_size  = next_position - position;
  segments_diff[segment_idx] = segment_size - segments_size_l;
}

template <typename VAL>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  extract_samples_segment(const VAL* data,
                          const size_t volume,
                          SegmentSample<VAL>* samples,
                          const size_t num_samples_per_segment_l,
                          const size_t segment_size_l,
                          const size_t offset,
                          const size_t num_sort_ranks,
                          const size_t sort_rank)
{
  const size_t sample_idx     = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t num_segments_l = volume / segment_size_l;
  const size_t num_samples_l  = num_samples_per_segment_l * num_segments_l;
  if (sample_idx >= num_samples_l) return;

  const size_t segment_id_l       = sample_idx / num_samples_per_segment_l;
  const size_t segment_sample_idx = sample_idx % num_samples_per_segment_l;
  const size_t sample_index       = offset + sample_idx;

  if (num_samples_per_segment_l < segment_size_l) {
    const size_t index = segment_id_l * segment_size_l +
                         (segment_sample_idx + 1) * segment_size_l / num_samples_per_segment_l - 1;
    samples[sample_index].value    = data[index];
    samples[sample_index].rank     = sort_rank;
    samples[sample_index].segment  = segment_id_l;
    samples[sample_index].position = index;
  } else {
    // edge case where num_samples_l > volume
    if (segment_sample_idx < segment_size_l) {
      const size_t index             = segment_id_l * segment_size_l + segment_sample_idx;
      samples[sample_index].value    = data[index];
      samples[sample_index].rank     = sort_rank;
      samples[sample_index].segment  = segment_id_l;
      samples[sample_index].position = index;
    } else {
      samples[sample_index].rank    = -1;  // not populated
      samples[sample_index].segment = segment_id_l;
    }
  }
}

struct subtract : public thrust::unary_function<int64_t, int64_t> {
  const int64_t constant_;

  subtract(int64_t constant) : constant_(constant) {}

  __host__ __device__ int64_t operator()(const int64_t& input) const { return input - constant_; }
};

struct positive_value : public thrust::unary_function<int64_t, int64_t> {
  __host__ __device__ int64_t operator()(const int64_t& x) const { return x > 0 ? x : 0; }
};

struct negative_value : public thrust::unary_function<int64_t, int64_t> {
  __host__ __device__ int64_t operator()(const int64_t& x) const { return x < 0 ? -x : 0; }
};

template <typename VAL>
SortPiece<VAL> sample_sort_nccl_nd(
  SortPiece<VAL> local_sorted,
  /* global domain information */
  size_t my_rank,  // global NCCL rank
  size_t num_ranks,
  size_t segment_size_g,
  /* domain information in sort dimension */
  size_t my_sort_rank,    // local rank id in sort dimension
  size_t num_sort_ranks,  // #ranks that share a sort dimension
  size_t* sort_ranks,     // rank ids that share a sort dimension with us
  size_t segment_size_l,  // (local) segment size
  /* other */
  bool argsort,
  hipStream_t stream,
  ncclComm_t* comm)
{
  size_t volume = local_sorted.size;

  /////////////////////////////////////////////////////////////////////////////////////////////////
  /////////////// Part 0: detection of empty nodes
  /////////////////////////////////////////////////////////////////////////////////////////////////

  // first of all we need to check for processes that don't want
  // to take part in the computation. This might lead to a reduction of
  // sort ranks. Note that if segment_size_l>0 && volume==0 means that we have
  // a full sort group being empty, this should not affect local sort rank size.
  {
    auto worker_count      = create_buffer<int32_t>(num_ranks, Memory::Z_COPY_MEM);
    auto* worker_count_ptr = worker_count.ptr(0);
    worker_count_ptr[0]    = segment_size_l > 0 ? 1 : 0;
    CHECK_NCCL(
      ncclAllReduce(worker_count_ptr, worker_count_ptr, 1, ncclInt32, ncclSum, *comm, stream));
    CHECK_CUDA(hipStreamSynchronize(stream));

    if (worker_count_ptr[0] < num_ranks) {
      const size_t number_sort_groups = num_ranks / num_sort_ranks;
      num_sort_ranks                  = worker_count_ptr[0] / number_sort_groups;
    }
    worker_count.destroy();

    // early out
    if (volume == 0) return local_sorted;
  }

  /////////////////////////////////////////////////////////////////////////////////////////////////
  /////////////// Part 1: select and share samples accross sort domain
  /////////////////////////////////////////////////////////////////////////////////////////////////

  // collect local samples - for now we take num_sort_ranks samples for every node/line
  // worst case this leads to imbalance of x2
  size_t num_segments_l            = volume / segment_size_l;
  size_t num_samples_per_segment_l = num_sort_ranks;
  size_t num_samples_l             = num_samples_per_segment_l * num_segments_l;
  size_t num_samples_per_segment_g = num_samples_per_segment_l * num_sort_ranks;
  size_t num_samples_g             = num_samples_per_segment_g * num_segments_l;
  auto samples = create_buffer<SegmentSample<VAL>>(num_samples_g, Memory::GPU_FB_MEM);

  size_t offset = num_samples_l * my_sort_rank;
  {
    const size_t num_blocks = (num_samples_l + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    extract_samples_segment<<<num_blocks, THREADS_PER_BLOCK, 0, stream>>>(
      local_sorted.values.ptr(0),
      volume,
      samples.ptr(0),
      num_samples_per_segment_l,
      segment_size_l,
      offset,
      num_sort_ranks,
      my_sort_rank);
  }

  // AllGather does not work here as not all have the same amount!
  // This is all2all restricted to one sort row
  {
    // allocate receive buffer
    const size_t aligned_count = get_16b_aligned_count(num_samples_l, sizeof(SegmentSample<VAL>));
    auto send_buffer = create_buffer<SegmentSample<VAL>>(aligned_count, Memory::GPU_FB_MEM);
    CHECK_CUDA(hipMemcpyAsync(send_buffer.ptr(0),
                               samples.ptr(offset),
                               sizeof(SegmentSample<VAL>) * num_samples_l,
                               hipMemcpyDeviceToDevice,
                               stream));

    auto recv_buffer =
      create_buffer<SegmentSample<VAL>>(aligned_count * num_sort_ranks, Memory::GPU_FB_MEM);

    CHECK_NCCL(ncclGroupStart());
    for (size_t r = 0; r < num_sort_ranks; r++) {
      if (r != my_sort_rank) {
        CHECK_NCCL(ncclSend(send_buffer.ptr(0),
                            num_samples_l * sizeof(SegmentSample<VAL>),
                            ncclInt8,
                            sort_ranks[r],
                            *comm,
                            stream));
        CHECK_NCCL(ncclRecv(recv_buffer.ptr(aligned_count * r),
                            num_samples_l * sizeof(SegmentSample<VAL>),
                            ncclInt8,
                            sort_ranks[r],
                            *comm,
                            stream));
      }
    }
    CHECK_NCCL(ncclGroupEnd());

    // copy back
    for (size_t r = 0; r < num_sort_ranks; r++) {
      if (r != my_sort_rank) {
        CHECK_CUDA(hipMemcpyAsync(samples.ptr(num_samples_l * r),
                                   recv_buffer.ptr(aligned_count * r),
                                   sizeof(SegmentSample<VAL>) * num_samples_l,
                                   hipMemcpyDeviceToDevice,
                                   stream));
      }
    }

    // destroy
    send_buffer.destroy();
    recv_buffer.destroy();
  }

  /////////////////////////////////////////////////////////////////////////////////////////////////
  /////////////// Part 2: select splitters from samples and collect positions in local data
  /////////////////////////////////////////////////////////////////////////////////////////////////

  // sort samples on device
  auto alloc       = ThrustAllocator(Memory::GPU_FB_MEM);
  auto exec_policy = thrust::cuda::par(alloc).on(stream);
  thrust::stable_sort(
    exec_policy, samples.ptr(0), samples.ptr(0) + num_samples_g, SegmentSampleComparator<VAL>());

  // check whether we have invalid samples (in case one participant did not have enough)
  SegmentSample<VAL> invalid_sample;
  invalid_sample.segment                = 0;
  invalid_sample.rank                   = -1;
  auto lower_bound                      = thrust::lower_bound(exec_policy,
                                         samples.ptr(0),
                                         samples.ptr(0) + num_samples_per_segment_g,
                                         invalid_sample,
                                         SegmentSampleComparator<VAL>());
  size_t num_usable_samples_per_segment = lower_bound - samples.ptr(0);

  // select splitters / positions based on samples (on device)
  // the indexing is split_positions[segments][positions]
  const size_t num_splitters = (num_sort_ranks - 1) * num_segments_l;
  auto split_positions       = create_buffer<size_t>(num_splitters, Memory::Z_COPY_MEM);
  {
    const size_t num_blocks = (num_splitters + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    extract_split_positions_segments<<<num_blocks, THREADS_PER_BLOCK, 0, stream>>>(
      local_sorted.values.ptr(0),
      segment_size_l,
      samples.ptr(0),
      num_segments_l,
      num_samples_per_segment_g,
      num_usable_samples_per_segment,
      split_positions.ptr(0),
      num_splitters,
      my_sort_rank);
  }

  // need to sync as we share values in between host/device
  CHECK_CUDA(hipStreamSynchronize(stream));

  // initialize sizes to send
  std::vector<Buffer<size_t>> size_send(num_sort_ranks);
  for (size_t r = 0; r < num_sort_ranks; r++) {
    size_send[r] =
      create_buffer<size_t>(num_segments_l + 1, Memory::Z_COPY_MEM);  // last element stores sum
    size_send[r][num_segments_l] = 0;
  }
  for (size_t segment = 0; segment < num_segments_l; ++segment) {
    size_t last_position = segment_size_l * segment;  // positions global
    for (size_t r = 0; r < num_sort_ranks - 1; r++) {
      size_t cur_position   = split_positions[segment * (num_sort_ranks - 1) + r];
      size_send[r][segment] = cur_position - last_position;
      size_send[r][num_segments_l] += size_send[r][segment];
      last_position = cur_position;
    }
    {
      // last rank separately
      const size_t size                      = (segment + 1) * segment_size_l - last_position;
      size_send[num_sort_ranks - 1][segment] = size;
      size_send[num_sort_ranks - 1][num_segments_l] += size;
    }
  }

  // cleanup intermediate data structures
  samples.destroy();
  split_positions.destroy();

  /////////////////////////////////////////////////////////////////////////////////////////////////
  /////////////// Part 3: communicate data in sort domain
  /////////////////////////////////////////////////////////////////////////////////////////////////

  // all2all exchange send/receive sizes
  std::vector<Buffer<size_t>> size_recv(num_sort_ranks);
  for (size_t r = 0; r < num_sort_ranks; r++) {
    size_recv[r] =
      create_buffer<size_t>(num_segments_l + 1, Memory::Z_COPY_MEM);  // last element stores sum
  }
  CHECK_NCCL(ncclGroupStart());
  for (size_t r = 0; r < num_sort_ranks; r++) {
    CHECK_NCCL(
      ncclSend(size_send[r].ptr(0), num_segments_l + 1, ncclUint64, sort_ranks[r], *comm, stream));
    CHECK_NCCL(
      ncclRecv(size_recv[r].ptr(0), num_segments_l + 1, ncclUint64, sort_ranks[r], *comm, stream));
  }
  CHECK_NCCL(ncclGroupEnd());

  // need to sync as we share values in between host/device
  CHECK_CUDA(hipStreamSynchronize(stream));

  // copy values into aligned send buffer
  std::vector<Buffer<VAL>> val_send_buffers(num_sort_ranks);
  std::vector<Buffer<size_t>> idc_send_buffers(num_sort_ranks);
  {
    std::vector<size_t> positions(num_sort_ranks);
    for (size_t r = 0; r < num_sort_ranks; r++) {
      val_send_buffers[r] = create_buffer<VAL>(size_send[r][num_segments_l], Memory::GPU_FB_MEM);
      if (argsort) {
        idc_send_buffers[r] =
          create_buffer<size_t>(size_send[r][num_segments_l], Memory::GPU_FB_MEM);
      }
      positions[r] = 0;
    }
    for (size_t segment = 0; segment < num_segments_l; ++segment) {
      size_t start_position = segment * segment_size_l;
      for (size_t r = 0; r < num_sort_ranks; r++) {
        size_t size = size_send[r][segment];
        CHECK_CUDA(hipMemcpyAsync(val_send_buffers[r].ptr(positions[r]),
                                   local_sorted.values.ptr(start_position),
                                   sizeof(VAL) * size,
                                   hipMemcpyDeviceToDevice,
                                   stream));
        if (argsort) {
          CHECK_CUDA(hipMemcpyAsync(idc_send_buffers[r].ptr(positions[r]),
                                     local_sorted.indices.ptr(start_position),
                                     sizeof(size_t) * size,
                                     hipMemcpyDeviceToDevice,
                                     stream));
        }
        start_position += size;
        positions[r] += size;

        assert(segment < num_segments_l - 1 || positions[r] == size_send[r][num_segments_l]);
      }
      assert(start_position == (segment + 1) * segment_size_l);
    }
    local_sorted.values.destroy();
    if (argsort) local_sorted.indices.destroy();
  }

  // allocate target buffers
  std::vector<SegmentMergePiece<VAL>> merge_buffers(num_sort_ranks);
  {
    for (size_t r = 0; r < num_sort_ranks; ++r) {
      auto size = size_recv[r][num_segments_l];

      merge_buffers[r].size     = size;
      merge_buffers[r].segments = create_buffer<size_t>(size, Memory::GPU_FB_MEM);

      // initialize segment information
      // TODO: maybe there is a better way?
      {
        // 0  1  2  1  3      // counts per segment to receive
        // 0  1  3  4  7
        // 0 1 2 3 4 5 6
        // 1 1 0 1 1 0 0
        // 1 2 2 3 4 4 4      // segment id for all received elements
        thrust::inclusive_scan(exec_policy,
                               size_recv[r].ptr(0),
                               size_recv[r].ptr(0) + num_segments_l + 1,
                               size_recv[r].ptr(0));
        CHECK_CUDA(hipMemsetAsync(merge_buffers[r].segments.ptr(0), 0, size * sizeof(size_t)));
        const size_t num_blocks = (num_segments_l + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        assert(sizeof(unsigned long long int) ==
               sizeof(size_t));  // kernel needs to cast for atomicAdd...
        initialize_segment_start_positions<<<num_blocks, THREADS_PER_BLOCK, 0, stream>>>(
          size_recv[r].ptr(0),
          num_segments_l - 1,
          merge_buffers[r].segments.ptr(0),
          merge_buffers[r].size);
        thrust::inclusive_scan(exec_policy,
                               merge_buffers[r].segments.ptr(0),
                               merge_buffers[r].segments.ptr(0) + size,
                               merge_buffers[r].segments.ptr(0));
        size_recv[r].destroy();
      }

      merge_buffers[r].values = create_buffer<VAL>(size, Memory::GPU_FB_MEM);
      if (argsort) {
        merge_buffers[r].indices = create_buffer<int64_t>(size, Memory::GPU_FB_MEM);
      } else {
        merge_buffers[r].indices = create_buffer<int64_t>(0, Memory::GPU_FB_MEM);
      }
    }
  }

  // communicate all2all (in sort dimension)
  CHECK_NCCL(ncclGroupStart());
  for (size_t r = 0; r < num_sort_ranks; r++) {
    CHECK_NCCL(ncclSend(val_send_buffers[r].ptr(0),
                        size_send[r][num_segments_l] * sizeof(VAL),
                        ncclInt8,
                        sort_ranks[r],
                        *comm,
                        stream));
    CHECK_NCCL(ncclRecv(merge_buffers[r].values.ptr(0),
                        merge_buffers[r].size * sizeof(VAL),
                        ncclInt8,
                        sort_ranks[r],
                        *comm,
                        stream));
  }
  CHECK_NCCL(ncclGroupEnd());

  if (argsort) {
    CHECK_NCCL(ncclGroupStart());
    for (size_t r = 0; r < num_sort_ranks; r++) {
      CHECK_NCCL(ncclSend(idc_send_buffers[r].ptr(0),
                          size_send[r][num_segments_l],
                          ncclInt64,
                          sort_ranks[r],
                          *comm,
                          stream));
      CHECK_NCCL(ncclRecv(merge_buffers[r].indices.ptr(0),
                          merge_buffers[r].size,
                          ncclInt64,
                          sort_ranks[r],
                          *comm,
                          stream));
    }
    CHECK_NCCL(ncclGroupEnd());
  }

  // cleanup remaining buffers
  for (size_t r = 0; r < num_sort_ranks; r++) {
    size_send[r].destroy();
    // size_recv[r].destroy(); has been released before
    val_send_buffers[r].destroy();
    if (argsort) idc_send_buffers[r].destroy();
  }

  /////////////////////////////////////////////////////////////////////////////////////////////////
  /////////////// Part 4: merge data
  /////////////////////////////////////////////////////////////////////////////////////////////////

  // now merge sort all into the result buffer
  // maybe k-way merge is more efficient here...
  for (size_t stride = 1; stride < num_sort_ranks; stride *= 2) {
    for (size_t pos = 0; pos + stride < num_sort_ranks; pos += 2 * stride) {
      SegmentMergePiece<VAL> source1 = merge_buffers[pos];
      SegmentMergePiece<VAL> source2 = merge_buffers[pos + stride];
      auto merged_size               = source1.size + source2.size;
      auto merged_values             = create_buffer<VAL>(merged_size);
      auto merged_segments           = create_buffer<size_t>(merged_size);
      auto merged_indices            = source1.indices;  // will be overriden for argsort
      auto p_merged_values           = merged_values.ptr(0);
      auto p_merged_segments         = merged_segments.ptr(0);
      auto p_values1                 = source1.values.ptr(0);
      auto p_values2                 = source2.values.ptr(0);
      auto p_segments1               = source1.segments.ptr(0);
      auto p_segments2               = source2.segments.ptr(0);

      auto comb_keys_1 = thrust::make_zip_iterator(thrust::make_tuple(p_segments1, p_values1));
      auto comb_keys_2 = thrust::make_zip_iterator(thrust::make_tuple(p_segments2, p_values2));
      auto comb_keys_merged =
        thrust::make_zip_iterator(thrust::make_tuple(p_merged_segments, p_merged_values));

      if (argsort) {
        merged_indices = create_buffer<int64_t>(merged_size);
        // merge with key/value
        auto p_indices1       = source1.indices.ptr(0);
        auto p_indices2       = source2.indices.ptr(0);
        auto p_merged_indices = merged_indices.ptr(0);
        thrust::merge_by_key(exec_policy,
                             comb_keys_1,
                             comb_keys_1 + source1.size,
                             comb_keys_2,
                             comb_keys_2 + source2.size,
                             p_indices1,
                             p_indices2,
                             comb_keys_merged,
                             p_merged_indices,
                             thrust::less<thrust::tuple<size_t, VAL>>());
        source1.indices.destroy();
      } else {
        thrust::merge(exec_policy,
                      comb_keys_1,
                      comb_keys_1 + source1.size,
                      comb_keys_2,
                      comb_keys_2 + source2.size,
                      comb_keys_merged,
                      thrust::less<thrust::tuple<size_t, VAL>>());
      }

      source1.values.destroy();
      source2.values.destroy();
      source1.segments.destroy();
      source2.segments.destroy();
      source2.indices.destroy();

      merge_buffers[pos].values   = merged_values;
      merge_buffers[pos].indices  = merged_indices;
      merge_buffers[pos].segments = merged_segments;
      merge_buffers[pos].size     = merged_size;
    }
  }

  /////////////////////////////////////////////////////////////////////////////////////////////////
  /////////////// Part 5: re-balance data to match input/output dimensions
  /////////////////////////////////////////////////////////////////////////////////////////////////

  // rebalance step
  SortPiece<VAL> result;
  {
    // compute diff for each segment
    auto segment_diff = create_buffer<int64_t>(num_segments_l, Memory::Z_COPY_MEM);
    {
      // start kernel to search from merge_buffers[0].segments
      const size_t num_blocks = (num_segments_l + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
      extract_segment_sizes<<<num_blocks, THREADS_PER_BLOCK, 0, stream>>>(
        merge_buffers[0].segments.ptr(0),
        merge_buffers[0].size,
        segment_diff.ptr(0),
        num_segments_l,
        segment_size_l);
    }

    // not needed anymore
    merge_buffers[0].segments.destroy();

    // allocate target
    std::vector<Buffer<int64_t>> segment_diff_buffers(num_sort_ranks);
    for (size_t r = 0; r < num_sort_ranks; r++) {
      segment_diff_buffers[r] = create_buffer<int64_t>(num_segments_l, Memory::GPU_FB_MEM);
    }

    // communicate segment diffs
    CHECK_NCCL(ncclGroupStart());
    for (size_t r = 0; r < num_sort_ranks; r++) {
      CHECK_NCCL(
        ncclSend(segment_diff.ptr(0), num_segments_l, ncclInt64, sort_ranks[r], *comm, stream));
      CHECK_NCCL(ncclRecv(
        segment_diff_buffers[r].ptr(0), num_segments_l, ncclInt64, sort_ranks[r], *comm, stream));
    }
    CHECK_NCCL(ncclGroupEnd());

    // copy to transpose structure [segments][ranks]
    auto segment_diff_2d =
      create_buffer<int64_t>(num_segments_l * num_sort_ranks, Memory::GPU_FB_MEM);
    for (size_t r = 0; r < num_sort_ranks; r++) {
      CHECK_CUDA(hipMemcpy2DAsync(segment_diff_2d.ptr(r),
                                   num_sort_ranks * sizeof(int64_t),
                                   segment_diff_buffers[r].ptr(0),
                                   1 * sizeof(int64_t),
                                   sizeof(int64_t),
                                   num_segments_l,
                                   hipMemcpyDeviceToDevice,
                                   stream));
    }
    for (size_t r = 0; r < num_sort_ranks; r++) { segment_diff_buffers[r].destroy(); }

#ifdef DEBUG_CUNUMERIC
    for (size_t segment = 0; segment < num_segments_l; ++segment) {
      size_t start_position = segment * num_sort_ranks;
      assert(0 == thrust::reduce(exec_policy,
                                 segment_diff_2d.ptr(start_position),
                                 segment_diff_2d.ptr(start_position) + num_sort_ranks,
                                 0));
    }
#endif

    // 2d data [segments][ranks]
    /*
          -2    2    1    1    -3     2    -1
          -2    0    1    2    -1     1     0  (inclusive scan)
          neg --> receive from right
          pos --> send right

          0    2    0    -1   -2    1    -1    (incl.scan right)
          neg --> receive from left
          pos --> send left

          edge case --> send more than whole line should not happen due to sample choice!
    */
    // 2 (signed) arrays - left/right for every segment
    auto send_left  = create_buffer<int64_t>(num_segments_l, Memory::Z_COPY_MEM);
    auto send_right = create_buffer<int64_t>(num_segments_l, Memory::Z_COPY_MEM);

    // compute data to send....
    auto segment_diff_2d_scan =
      create_buffer<int64_t>(num_segments_l * num_sort_ranks, Memory::GPU_FB_MEM);
    thrust::device_ptr<int64_t> segment_diff_2d_ptr(segment_diff_2d.ptr(0));
    thrust::device_ptr<int64_t> segment_diff_2d_scan_ptr(segment_diff_2d_scan.ptr(0));
    thrust::inclusive_scan(exec_policy,
                           segment_diff_2d_ptr,
                           segment_diff_2d_ptr + num_segments_l * num_sort_ranks,
                           segment_diff_2d_scan_ptr);
    CHECK_CUDA(hipMemcpy2DAsync(send_right.ptr(0),
                                 sizeof(int64_t),
                                 segment_diff_2d_scan.ptr(0) + my_sort_rank,
                                 num_sort_ranks * sizeof(int64_t),
                                 sizeof(int64_t),
                                 num_segments_l,
                                 hipMemcpyDeviceToDevice,
                                 stream));
    thrust::reverse_iterator<thrust::device_vector<int64_t>::iterator> iter_in(
      segment_diff_2d_ptr + num_segments_l * num_sort_ranks);
    thrust::reverse_iterator<thrust::device_vector<int64_t>::iterator> iter_out(
      segment_diff_2d_scan_ptr + num_segments_l * num_sort_ranks);
    thrust::inclusive_scan(
      exec_policy, iter_in, iter_in + num_segments_l * num_sort_ranks, iter_out);
    CHECK_CUDA(hipMemcpy2DAsync(send_left.ptr(0),
                                 sizeof(int64_t),
                                 segment_diff_2d_scan.ptr(0) + my_sort_rank,
                                 num_sort_ranks * sizeof(int64_t),
                                 sizeof(int64_t),
                                 num_segments_l,
                                 hipMemcpyDeviceToDevice,
                                 stream));
    segment_diff_2d.destroy();
    segment_diff_2d_scan.destroy();

    // package data to send
    size_t send_left_size  = thrust::transform_reduce(exec_policy,
                                                     send_left.ptr(0),
                                                     send_left.ptr(0) + num_segments_l,
                                                     positive_value(),
                                                     0,
                                                     thrust::plus<int64_t>());
    size_t recv_left_size  = thrust::transform_reduce(exec_policy,
                                                     send_left.ptr(0),
                                                     send_left.ptr(0) + num_segments_l,
                                                     negative_value(),
                                                     0,
                                                     thrust::plus<int64_t>());
    size_t send_right_size = thrust::transform_reduce(exec_policy,
                                                      send_right.ptr(0),
                                                      send_right.ptr(0) + num_segments_l,
                                                      positive_value(),
                                                      0,
                                                      thrust::plus<int64_t>());
    size_t recv_right_size = thrust::transform_reduce(exec_policy,
                                                      send_right.ptr(0),
                                                      send_right.ptr(0) + num_segments_l,
                                                      negative_value(),
                                                      0,
                                                      thrust::plus<int64_t>());
    SortPiece<VAL> send_left_data, recv_left_data, send_right_data, recv_right_data;
    send_left_data.values  = create_buffer<VAL>(send_left_size, Memory::GPU_FB_MEM);
    recv_left_data.values  = create_buffer<VAL>(recv_left_size, Memory::GPU_FB_MEM);
    send_right_data.values = create_buffer<VAL>(send_right_size, Memory::GPU_FB_MEM);
    recv_right_data.values = create_buffer<VAL>(recv_right_size, Memory::GPU_FB_MEM);
    send_left_data.size    = 0;  // will be incremented when data is inserted
    recv_left_data.size    = recv_left_size;
    send_right_data.size   = 0;  // will be incremented when data is inserted
    recv_right_data.size   = recv_right_size;
    if (argsort) {
      send_left_data.indices  = create_buffer<int64_t>(send_left_size, Memory::GPU_FB_MEM);
      recv_left_data.indices  = create_buffer<int64_t>(recv_left_size, Memory::GPU_FB_MEM);
      send_right_data.indices = create_buffer<int64_t>(send_right_size, Memory::GPU_FB_MEM);
      recv_right_data.indices = create_buffer<int64_t>(recv_right_size, Memory::GPU_FB_MEM);
    }
    auto segment_diff_ptr = segment_diff.ptr(0);
    auto send_left_ptr    = send_left.ptr(0);
    auto send_right_ptr   = send_right.ptr(0);
    {
      size_t start_pos = 0;
      for (size_t segment = 0; segment < num_segments_l; ++segment) {
        size_t end_pos = start_pos + segment_size_l + segment_diff_ptr[segment];
        if (send_left_ptr[segment] > 0) {
          auto size = send_left_ptr[segment];
          CHECK_CUDA(hipMemcpyAsync(send_left_data.values.ptr(send_left_data.size),
                                     merge_buffers[0].values.ptr(start_pos),
                                     sizeof(VAL) * size,
                                     hipMemcpyDeviceToDevice,
                                     stream));
          if (argsort) {
            CHECK_CUDA(hipMemcpyAsync(send_left_data.indices.ptr(send_left_data.size),
                                       merge_buffers[0].indices.ptr(start_pos),
                                       sizeof(int64_t) * size,
                                       hipMemcpyDeviceToDevice,
                                       stream));
          }
          send_left_data.size += size;
        }
        if (send_right_ptr[segment] > 0) {
          auto size = send_right_ptr[segment];
          CHECK_CUDA(hipMemcpyAsync(send_right_data.values.ptr(send_right_data.size),
                                     merge_buffers[0].values.ptr(end_pos - size),
                                     sizeof(VAL) * size,
                                     hipMemcpyDeviceToDevice,
                                     stream));
          if (argsort) {
            CHECK_CUDA(hipMemcpyAsync(send_right_data.indices.ptr(send_right_data.size),
                                       merge_buffers[0].indices.ptr(end_pos - size),
                                       sizeof(int64_t) * size,
                                       hipMemcpyDeviceToDevice,
                                       stream));
          }
          send_right_data.size += size;
        }
        start_pos = end_pos;
      }
      assert(send_left_data.size == send_left_size);
      assert(send_right_data.size == send_right_size);
    }

    // send/receive overlapping data
    {
      CHECK_NCCL(ncclGroupStart());
      if (send_left_size > 0) {
        CHECK_NCCL(ncclSend(send_left_data.values.ptr(0),
                            send_left_data.size * sizeof(VAL),
                            ncclInt8,
                            sort_ranks[my_sort_rank - 1],
                            *comm,
                            stream));
      }
      if (send_right_size > 0) {
        CHECK_NCCL(ncclSend(send_right_data.values.ptr(0),
                            send_right_data.size * sizeof(VAL),
                            ncclInt8,
                            sort_ranks[my_sort_rank + 1],
                            *comm,
                            stream));
      }
      if (recv_left_size > 0) {
        CHECK_NCCL(ncclRecv(recv_left_data.values.ptr(0),
                            recv_left_data.size * sizeof(VAL),
                            ncclInt8,
                            sort_ranks[my_sort_rank - 1],
                            *comm,
                            stream));
      }
      if (recv_right_size > 0) {
        CHECK_NCCL(ncclRecv(recv_right_data.values.ptr(0),
                            recv_right_data.size * sizeof(VAL),
                            ncclInt8,
                            sort_ranks[my_sort_rank + 1],
                            *comm,
                            stream));
      }

      if (argsort) {
        if (send_left_size > 0) {
          CHECK_NCCL(ncclSend(send_left_data.indices.ptr(0),
                              send_left_data.size,
                              ncclInt64,
                              sort_ranks[my_sort_rank - 1],
                              *comm,
                              stream));
        }
        if (send_right_size > 0) {
          CHECK_NCCL(ncclSend(send_right_data.indices.ptr(0),
                              send_right_data.size,
                              ncclInt64,
                              sort_ranks[my_sort_rank + 1],
                              *comm,
                              stream));
        }
        if (recv_left_size > 0) {
          CHECK_NCCL(ncclRecv(recv_left_data.indices.ptr(0),
                              recv_left_data.size,
                              ncclInt64,
                              sort_ranks[my_sort_rank - 1],
                              *comm,
                              stream));
        }
        if (recv_right_size > 0) {
          CHECK_NCCL(ncclRecv(recv_right_data.indices.ptr(0),
                              recv_right_data.size,
                              ncclInt64,
                              sort_ranks[my_sort_rank + 1],
                              *comm,
                              stream));
        }
      }
      CHECK_NCCL(ncclGroupEnd());

      send_left_data.values.destroy();
      send_right_data.values.destroy();
      if (argsort) {
        send_left_data.indices.destroy();
        send_right_data.indices.destroy();
      }
    }

    // merge data into target
    result.size   = volume;
    result.values = create_buffer<VAL>(volume, Memory::GPU_FB_MEM);
    if (argsort) { result.indices = create_buffer<int64_t>(volume, Memory::GPU_FB_MEM); }

    {
      size_t start_pos      = 0;
      size_t result_pos     = 0;
      size_t left_read_pos  = 0;
      size_t right_read_pos = 0;
      for (size_t segment = 0; segment < num_segments_l; ++segment) {
        size_t end_pos = start_pos + segment_size_l + segment_diff_ptr[segment];

        size_t copy_start = start_pos;
        size_t copy_end   = end_pos;

        if (send_left_ptr[segment] < 0) {
          // we have data to merge
          size_t received_size = -send_left_ptr[segment];
          CHECK_CUDA(hipMemcpyAsync(result.values.ptr(result_pos),
                                     recv_left_data.values.ptr(left_read_pos),
                                     sizeof(VAL) * received_size,
                                     hipMemcpyDeviceToDevice,
                                     stream));
          if (argsort)
            CHECK_CUDA(hipMemcpyAsync(result.indices.ptr(result_pos),
                                       recv_left_data.indices.ptr(left_read_pos),
                                       sizeof(int64_t) * received_size,
                                       hipMemcpyDeviceToDevice,
                                       stream));
          result_pos += received_size;
          left_read_pos += received_size;
        }

        // assemble line from old data and received data
        if (send_left_ptr[segment] > 0) copy_start += send_left_ptr[segment];
        if (send_right_ptr[segment] > 0) copy_end -= send_right_ptr[segment];
        {
          CHECK_CUDA(hipMemcpyAsync(result.values.ptr(result_pos),
                                     merge_buffers[0].values.ptr(copy_start),
                                     sizeof(VAL) * (copy_end - copy_start),
                                     hipMemcpyDeviceToDevice,
                                     stream));
          if (argsort)
            CHECK_CUDA(hipMemcpyAsync(result.indices.ptr(result_pos),
                                       merge_buffers[0].indices.ptr(copy_start),
                                       sizeof(int64_t) * (copy_end - copy_start),
                                       hipMemcpyDeviceToDevice,
                                       stream));
          result_pos += (copy_end - copy_start);
        }

        if (send_right_ptr[segment] < 0) {
          // we have data to merge
          size_t received_size = -send_right_ptr[segment];
          CHECK_CUDA(hipMemcpyAsync(result.values.ptr(result_pos),
                                     recv_right_data.values.ptr(right_read_pos),
                                     sizeof(VAL) * received_size,
                                     hipMemcpyDeviceToDevice,
                                     stream));
          if (argsort)
            CHECK_CUDA(hipMemcpyAsync(result.indices.ptr(result_pos),
                                       recv_right_data.indices.ptr(right_read_pos),
                                       sizeof(int64_t) * received_size,
                                       hipMemcpyDeviceToDevice,
                                       stream));
          result_pos += received_size;
          right_read_pos += received_size;
        }

        assert(result_pos == (segment + 1) * segment_size_l);
        start_pos = end_pos;
      }
    }

    // remove segment_sizes, all buffers should be destroyed...
    segment_diff.destroy();
    send_left.destroy();
    send_right.destroy();
    merge_buffers[0].values.destroy();
    recv_left_data.values.destroy();
    recv_right_data.values.destroy();
    if (argsort) {
      merge_buffers[0].indices.destroy();
      recv_left_data.indices.destroy();
      recv_right_data.indices.destroy();
    }
  }

  return result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////

template <LegateTypeCode CODE, int32_t DIM>
struct SortImplBody<VariantKind::GPU, CODE, DIM> {
  using VAL = legate_type_of<CODE>;

  void operator()(const Array& input_array,
                  Array& output_array,
                  const Pitches<DIM - 1>& pitches,
                  const Rect<DIM>& rect,
                  const size_t volume,
                  const size_t segment_size_l,
                  const size_t segment_size_g,
                  const bool argsort,
                  const bool stable,
                  const bool is_index_space,
                  const size_t local_rank,
                  const size_t num_ranks,
                  const size_t num_sort_ranks,
                  const std::vector<comm::Communicator>& comms)
  {
    auto input = input_array.read_accessor<VAL, DIM>(rect);

    // we allow empty domains for distributed sorting
    assert(rect.empty() || input.accessor.is_dense_row_major(rect));

    auto stream                = get_cached_stream();
    bool need_distributed_sort = segment_size_l != segment_size_g;

    // initialize sort pointers
    SortPiece<VAL> local_sorted;
    int64_t* indices_ptr = nullptr;
    VAL* values_ptr      = nullptr;
    if (argsort) {
      // make a buffer for input
      auto input_copy     = create_buffer<VAL>(volume, Legion::Memory::Kind::GPU_FB_MEM);
      local_sorted.values = input_copy;
      values_ptr          = input_copy.ptr(0);

      // initialize indices
      if (need_distributed_sort || output_array.dim() == -1) {
        auto indices_buffer  = create_buffer<int64_t>(volume, Legion::Memory::Kind::GPU_FB_MEM);
        indices_ptr          = indices_buffer.ptr(0);
        local_sorted.indices = indices_buffer;
        local_sorted.size    = volume;
      } else {
        AccessorWO<int64_t, DIM> output = output_array.write_accessor<int64_t, DIM>(rect);
        assert(output.accessor.is_dense_row_major(rect));
        indices_ptr = output.ptr(rect.lo);
      }
      size_t offset = rect.lo[DIM - 1];
      if (volume > 0) {
        if (DIM == 1) {
          thrust::sequence(thrust::cuda::par.on(stream), indices_ptr, indices_ptr + volume, offset);
        } else {
          thrust::transform(thrust::cuda::par.on(stream),
                            thrust::make_counting_iterator<int64_t>(0),
                            thrust::make_counting_iterator<int64_t>(volume),
                            thrust::make_constant_iterator<int64_t>(segment_size_l),
                            indices_ptr,
                            modulusWithOffset(offset));
        }
      }
    } else {
      // initialize output
      if (need_distributed_sort || output_array.dim() == -1) {
        auto input_copy      = create_buffer<VAL>(volume, Legion::Memory::Kind::GPU_FB_MEM);
        values_ptr           = input_copy.ptr(0);
        local_sorted.values  = input_copy;
        local_sorted.indices = create_buffer<int64_t>(0, Legion::Memory::Kind::GPU_FB_MEM);
        local_sorted.size    = volume;
      } else {
        AccessorWO<VAL, DIM> output = output_array.write_accessor<VAL, DIM>(rect);
        assert(output.accessor.is_dense_row_major(rect));
        values_ptr = output.ptr(rect.lo);
      }
    }
    if (volume > 0) {
      // sort data (locally)
      local_sort<CODE>(input.ptr(rect.lo),
                       values_ptr,
                       indices_ptr,
                       indices_ptr,
                       volume,
                       segment_size_l,
                       stable,
                       stream);
    }

    // this is linked to the decision in sorting.py on when to use an 'unbounded' output array.
    if (output_array.dim() == -1) {
      assert(DIM == 1);
      SortPiece<VAL> local_sorted_repartitioned =
        is_index_space
          ? sample_sort_nccl(
              local_sorted, local_rank, num_ranks, argsort, stream, comms[0].get<ncclComm_t*>())
          : local_sorted;
      if (argsort) {
        output_array.return_data(local_sorted_repartitioned.indices,
                                 local_sorted_repartitioned.size);
      } else {
        output_array.return_data(local_sorted_repartitioned.values,
                                 local_sorted_repartitioned.size);
      }
    } else {
      if (need_distributed_sort) {
        assert(DIM > 1);
        assert(is_index_space);
        std::vector<size_t> sort_ranks(num_sort_ranks);
        size_t rank_group = local_rank / num_sort_ranks;
        for (int r = 0; r < num_sort_ranks; ++r) sort_ranks[r] = rank_group * num_sort_ranks + r;
        SortPiece<VAL> final_sorted_flattened = sample_sort_nccl_nd(local_sorted,
                                                                    local_rank,
                                                                    num_ranks,
                                                                    segment_size_g,
                                                                    local_rank % num_sort_ranks,
                                                                    num_sort_ranks,
                                                                    &sort_ranks[0],
                                                                    segment_size_l,
                                                                    argsort,
                                                                    stream,
                                                                    comms[0].get<ncclComm_t*>());
        assert(final_sorted_flattened.size == volume);

        if (argsort) {
          auto output = output_array.write_accessor<int64_t, DIM>(rect);
          CHECK_CUDA(hipMemcpyAsync(output.ptr(rect.lo),
                                     final_sorted_flattened.indices.ptr(0),
                                     sizeof(int64_t) * volume,
                                     hipMemcpyDeviceToDevice,
                                     stream));
          final_sorted_flattened.indices.destroy();
        } else {
          auto output = output_array.write_accessor<VAL, DIM>(rect);
          CHECK_CUDA(hipMemcpyAsync(output.ptr(rect.lo),
                                     final_sorted_flattened.values.ptr(0),
                                     sizeof(VAL) * volume,
                                     hipMemcpyDeviceToDevice,
                                     stream));
        }
        final_sorted_flattened.values.destroy();
      } else if (argsort) {
        // cleanup
        local_sorted.values.destroy();
      }
    }
  }
};

/*static*/ void SortTask::gpu_variant(TaskContext& context)
{
  sort_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
